
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void c_hello() {
	printf("Hello World from the GPU! (ThrIndex:%d)\n", threadIdx.x);
}

int main() {
	c_hello <<<1,10>>>();
	hipDeviceSynchronize();
	//Needed for output.
	return 0;
}