#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <time.h>
#include <string.h>
#include "omp.h"
#include "matrix.h"

#define DEBUG false
#define CUDA true
#define NUM_TESTS 5
#define MAX_THREADS 2014
#define MAX_BLOCKS 1024

clock_t start, end;

void runSerial(const MATRIX mA, const MATRIX mB, MATRIX* mC, double* times) {
	clock_t start, end;
	int i = 0;

	double totalTime;
	for (; i < NUM_TESTS; i++) {
		start = clock();
		multiplyMatrix(
		/* startPos */ 	0,
		/* endPos */ 	mC->rows * mC->cols,
		/* matrix A */ 	mA,
		/* matrix B */ 	mB,
		/* matrix C */ 	mC);
    	end = clock();

    	totalTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    	*(times + i) = totalTime;
    	memset(mC->vals, 0, (mC->rows * mC->cols)*sizeof(double));
	}
}

void runOmp(MATRIX* mA, MATRIX* mB, MATRIX* mC, double* times) {
	clock_t start, end;
	int i = 0;

	int totalWork = mC->rows * mC->cols;

	int totalThreads = omp_get_max_threads();
	if (totalThreads > totalWork) {
		totalThreads = totalWork;
	}

	int workPerThread = totalWork / totalThreads;

	double totalTime;
	for (; i < NUM_TESTS; i++) {
		start = clock();
		#pragma omp parallel num_threads(totalThreads) shared(workPerThread, mA, mB, mC)
		{
			int startPos = omp_get_thread_num() * workPerThread;
			int endPos = startPos + workPerThread;

			multiplyMatrix(
				/* startPos */	startPos,
				/* endPos */	endPos,
				/* matrix A */	*mA,
				/* matrix B */	*mB,
				/* matrix C */	mC);
		}
    	end = clock();

    	totalTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    	*(times + i) = totalTime;
    	memset(mC->vals, 0, (mC->rows * mC->cols)*sizeof(double));
	}
}

__global__ void calculateMatrixCuda(int *workPerThread, MATRIX* mA, MATRIX* mB, MATRIX* mC) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate index for each thread
	int pos = idx * *workPerThread;
	int endPos = pos + *workPerThread;

	int n = mA->cols;
	for (; pos < mC->rows * mC->cols && pos < endPos; pos++) {
		int row = pos / mC->rows;
		int col = pos % mC->rows;

		double sum = 0.0;
		int i = 0;
		for (; i < n; i++) {
			sum += (*(mA->vals + mA->cols * row + i) * *(mB->vals + mB->cols * col + i));
		}

		*(mC->vals + pos) = sum;
	}
}

void runCuda(MATRIX* mA, MATRIX* mB, MATRIX* mC, double* times) {
	clock_t start, end;
	int i = 0;

	int *workPerThread;
	hipMallocManaged(&workPerThread, sizeof(int));

	int totalBlocks = mC->rows < MAX_BLOCKS ?  mC->rows : MAX_BLOCKS;
	int totalRows = mC->cols < MAX_THREADS ?  mC->cols : MAX_THREADS;

	int totalWork = mC->rows * mC->cols;
	*workPerThread = totalWork / (totalBlocks * totalRows);

	double totalTime;
	for (; i < NUM_TESTS; i++) {
		start = clock();

		calculateMatrixCuda <<<totalBlocks, totalRows>>> (workPerThread, mA, mB, mC);
		hipDeviceSynchronize();

    	end = clock();

    	totalTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    	*(times + i) = totalTime;
    	printMatrix(*mC, 'C');
    	memset(mC->vals, 0, (mC->rows * mC->cols)*sizeof(double));
	}
}

int main(int argc, char *argv[]) {
	if (!verifyArgs(argc))
		return false;

	MATRIX *mA, *mB;
	if (!initializeInputMatrixes(argc, argv, &mA, &mB, DEBUG, CUDA))
		return -1;

	MATRIX* mC;
	if  ((mC = initializeOutputMatrix(*mA, *mB, CUDA)) == NULL) {
		printf("Error allocating output matrix C.\n");
		return false;
	}

	MATRIX *mBT = transposeMatrix(*mB, CUDA);

	freeMatrix(mB, CUDA);

	mB = mBT;

	if (DEBUG) {
		printf("Matrix B Transposed: \n");
		printMatrix(*mBT, 'T');
	}

	double *serialTimes = (double *)malloc(NUM_TESTS * sizeof(double));
	runSerial(*mA, *mB, mC, serialTimes);

	double *ompTimes = (double *)malloc(NUM_TESTS * sizeof(double));
	runOmp(mA, mB, mC, ompTimes);

	double *cudaTimes = (double *)malloc(NUM_TESTS * sizeof(double));
	runCuda(mA, mB, mC, cudaTimes);

	freeMatrix(mA, CUDA);
	freeMatrix(mB, CUDA);
	freeMatrix(mC, CUDA);

	printf("%20s %20s %20s\n", "SERIAL", "OMP", "CUDA");

	int i;
	for (i = 0; i < NUM_TESTS; i++)
		printf("%20lf %20lf %20lf\n", *(serialTimes + i), *(ompTimes + i), *(cudaTimes + i));

	printf("\n");

	return 0;
}