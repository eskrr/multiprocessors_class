#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <time.h>
#include "matrix.h"

#define DEBUG true
#define MAX_THREADS 2014
#define MAX_BLOCKS 1024

clock_t start, end;

void printDeviceInfo() {
	int nDevices;
	hipGetDeviceCount(&nDevices);
	int i = 0;
	for(i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Maximum number of 32-bit registers: %d\n", prop.regsPerBlock);
		printf("  Maximum number of threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("  Maximum block dimension: [%d,%d,%d]\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("  Maximum grid size: [%d,%d,%d]\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
		printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	}
}

__global__ void calculateMatrixCuda(int *workPerThread, MATRIX* mA, MATRIX* mB, MATRIX* mC) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate index for each thread
	int startPos = idx * *workPerThread;
	int endPos = startPos + *workPerThread;

	printf("(ThreadId: %d, WorkPerThread: %d)\n", idx, *workPerThread);
	printf("(start: %d, end: %d)\n", startPos, endPos);
	printf("mA: rows: %d, cols: %d\n", mA->rows, mA->cols);
	printf("mB: rows: %d, cols: %d\n", mB->rows, mB->cols);
	printf("mC: rows: %d, cols: %d\n", mC->rows, mC->cols);
}

int main(int argc, char *argv[]) {
	if (DEBUG)
		printDeviceInfo();

	if (!verifyArgs(argc))
		return false;

	MATRIX *mA, *mB;
	if (!initializeInputMatrixes(argc, argv, &mA, &mB, DEBUG, true))
		printf("Error allocating input matrixes.\n");
		return -1;

	printf("Inpit Matrix allocated\n");

	MATRIX* mC;

	if  ((mC = initializeOutputMatrix(*mA, *mB, true)) == NULL) {
		printf("Error allocating output matrix C.\n");
		return -1;
	}

	printf("Que pedo?\n");

	int *workPerThread;
	hipMallocManaged(&workPerThread, sizeof(int));
	printf("Que pedo2?\n");

	int totalBlocks = mC->rows < MAX_BLOCKS ?  mC->rows : MAX_BLOCKS;
	int totalRows = mC->cols < MAX_THREADS ?  mC->cols : MAX_THREADS;
	printf("Que pedo4?\n");

	int totalWork = mC->rows * mC->cols;
	*workPerThread = totalWork / (totalBlocks * totalRows);

	printf("totalBlocks: %d, totalRows: %d\n", totalBlocks, totalRows);

	start = clock();
	calculateMatrixCuda <<<totalBlocks, totalRows>>> (workPerThread, mA, mB, mC);
	hipDeviceSynchronize();
	end = clock();

 	// double totalTime = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Total time taken by CPU: %lf\n", end - start); 

	printf("Verifying matrix... \n");
	//Needed for output.
	return 0;
}