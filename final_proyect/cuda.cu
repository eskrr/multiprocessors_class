#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <time.h>
#include "matrix.h"

#define DEBUG true
#define MAX_THREADS 2014
#define MAX_BLOCKS 1024

clock_t start, end;

void printDeviceInfo() {
	int nDevices;
	hipGetDeviceCount(&nDevices);
	int i = 0;
	for(i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Maximum number of 32-bit registers: %d\n", prop.regsPerBlock);
		printf("  Maximum number of threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("  Maximum block dimension: [%d,%d,%d]\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("  Maximum grid size: [%d,%d,%d]\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
		printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	}
}

__global__ void calculateMatrixCuda(int *workPerThread) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate index for each thread
	int startPos = idx * *workPerThread;
	int endPos = startPos + *workPerThread;

	printf("(ThreadId: %d, WorkPerThread: %d)\n", idx, *workPerThread);
	printf("(start: %d, end: %d)\n", startPos, endPos);
}

int main(int argc, char *argv[]) {
	if (DEBUG)
		printDeviceInfo();

	if (!verifyArgs(argc))
		return false;

	MATRIX *mA, *mB;
	if (!initializeInputMatrixes(argc, argv, &mA, &mB, DEBUG, true))
		printf("Error allocating input matrixes.\n");
		return -1;

	MATRIX* mC;

	if  ((mC = initializeOutputMatrix(*mA, *mB, true)) == NULL) {
		printf("Error allocating output matrix C.\n");
		return -1;
	}

	int *workPerThread;
	hipMallocManaged(&workPerThread, sizeof(int));

	int totalBlocks = mC->rows < MAX_BLOCKS ?  mC->rows : MAX_BLOCKS;
	int totalRows = mC->cols < MAX_THREADS ?  mC->cols : MAX_THREADS;

	int totalWork = mC->rows * mC->cols;
	*workPerThread = totalWork / (totalBlocks * totalRows);

	start = clock();
	calculateMatrixCuda <<<totalBlocks, totalRows>>> (workPerThread);
	hipDeviceSynchronize();
	end = clock();

 	// double totalTime = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Total time taken by CPU: %lf\n", end - start); 

	printf("Verifying matrix... \n");
	//Needed for output.
	return 0;
}